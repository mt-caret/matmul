#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <stdexcept>
#include <vector>

#include "helper_cuda.hpp"

using namespace std;

vector<float> create_random_matrix(int n) {
  vector<float> matrix(n * n);
  for (int i = 0; i < n * n; ++i) {
    matrix[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
  }
  return matrix;
}

void cublas_matrix_multiply(hipblasHandle_t handle, float *d_m1, float *d_m2,
                            float *d_result, int n) {
  const float alpha = 1.0f;
  const float beta = 0.0f;

  // Run CUDA kernel. We swap the ordering of m1 and m2, which are in row-major
  // order, which on computation in column-major order (which cublas assumes)
  // will result in the correct result when interpreted in row-major order.
  // c.f. https://stackoverflow.com/a/56064726
  checkCublasErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                                &alpha, d_m2, n, d_m1, n, &beta, d_result, n));
  checkCudaErrors(hipDeviceSynchronize());
}

vector<float> cublas_matrix_multiply_and_return(const vector<float> &m1,
                                                const vector<float> &m2,
                                                int n) {
  float *d_m1, *d_m2, *d_result;
  hipblasHandle_t handle;
  checkCublasErrors(hipblasCreate(&handle));

  checkCudaErrors(hipMalloc(&d_m1, n * n * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_m2, n * n * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_result, n * n * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_m1, m1.data(), n * n * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2.data(), n * n * sizeof(float),
                             hipMemcpyHostToDevice));

  vector<float> cuda_result(n * n);

  cublas_matrix_multiply(handle, d_m1, d_m2, d_result, n);

  // Copy result back to host
  checkCudaErrors(hipMemcpy(cuda_result.data(), d_result,
                             n * n * sizeof(float), hipMemcpyDeviceToHost));

  hipblasDestroy(handle);
  return cuda_result;
}

void diff_with_cublas(const vector<float> &m1, const vector<float> &m2, int n,
                      const vector<float> result, string name) {
  vector<float> cublas_result = cublas_matrix_multiply_and_return(m1, m2, n);

  const float epsilon = 1e-2f;
  for (int i = 0; i < n * n; ++i) {
    if (std::abs(cublas_result[i] - result[i]) > epsilon) {
      cerr << "Significant mismatch at index " << i
           << ": cublas = " << cublas_result[i] << ", " << name << " = "
           << result[i] << '\n';
      throw runtime_error("Significant mismatch between blas and " + name +
                          " implementations");
    }
  }
}

__global__ void naive_matrix_multiply_kernel(float *m1, float *m2,
                                             float *result, int n) {
  // Note x and y are swapped; this causes less coalescing within a warp,
  // slowing the kernel down compared to naive_matrix_multiply_kernel2
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y < n && x < n) {
    float sum = 0;
    for (int k = 0; k < n; ++k) {
      sum += m1[y * n + k] * m2[k * n + x];
    }
    result[y * n + x] = sum;
  }
}

void measure_naive(const vector<float> &m1, const vector<float> &m2, int size,
                   int runs) {
  // Transfer matrices to device
  float *d_m1, *d_m2, *d_result;
  checkCudaErrors(hipMalloc(&d_m1, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_m2, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_result, size * size * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_m1, m1.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));

  // Define grid and block dimensions
  dim3 blockDim(32, 32);
  dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
               (size + blockDim.y - 1) / blockDim.y);

  if (size < 5000) {
    vector<float> result(size * size);

    // Run CUDA kernel
    naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                        size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    checkCudaErrors(hipMemcpy(result.data(), d_result,
                               size * size * sizeof(float),
                               hipMemcpyDeviceToHost));

    diff_with_cublas(m1, m2, size, result, "naive_cuda");
  }

  // Warmup
  for (int i = 0; i < 10; ++i) {
    naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                        size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }

  for (int i = 0; i < runs; ++i) {
    auto start_time = chrono::high_resolution_clock::now();
    naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                        size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto end_time = chrono::high_resolution_clock::now();
    auto duration =
        chrono::duration_cast<chrono::microseconds>(end_time - start_time)
            .count();
    cout << size << "," << i + 1 << "," << duration << ",naive_cuda" << endl;
  }

  // Clean up
  checkCudaErrors(hipFree(d_m1));
  checkCudaErrors(hipFree(d_m2));
  checkCudaErrors(hipFree(d_result));
}

__global__ void naive_matrix_multiply_kernel2(float *m1, float *m2,
                                              float *result, int n) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (y < n && x < n) {
    float sum = 0;
    for (int k = 0; k < n; ++k) {
      sum += m1[y * n + k] * m2[k * n + x];
    }
    result[y * n + x] = sum;
  }
}

void measure_naive2(const vector<float> &m1, const vector<float> &m2, int size,
                    int runs) {
  // Transfer matrices to device
  float *d_m1, *d_m2, *d_result;
  checkCudaErrors(hipMalloc(&d_m1, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_m2, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_result, size * size * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_m1, m1.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));

  // Define grid and block dimensions
  dim3 blockDim(32, 32);
  dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
               (size + blockDim.y - 1) / blockDim.y);

  if (size < 5000) {
    vector<float> result(size * size);

    naive_matrix_multiply_kernel2<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                         size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(result.data(), d_result,
                               size * size * sizeof(float),
                               hipMemcpyDeviceToHost));

    diff_with_cublas(m1, m2, size, result, "naive_cuda2");
  }

  // Warmup
  for (int i = 0; i < 10; ++i) {
    naive_matrix_multiply_kernel2<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                         size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }

  for (int i = 0; i < runs; ++i) {
    auto start_time = chrono::high_resolution_clock::now();
    naive_matrix_multiply_kernel2<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                         size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto end_time = chrono::high_resolution_clock::now();
    auto duration =
        chrono::duration_cast<chrono::microseconds>(end_time - start_time)
            .count();
    cout << size << "," << i + 1 << "," << duration << ",naive_cuda2" << endl;
  }

  // Clean up
  checkCudaErrors(hipFree(d_m1));
  checkCudaErrors(hipFree(d_m2));
  checkCudaErrors(hipFree(d_result));
}

extern __shared__ float shared[];
__global__ void tiled_matrix_multiply_kernel(float *m1, float *m2,
                                             float *result, int n,
                                             size_t tile_size) {
  float *shared_m1 = shared;
  float *shared_m2 = shared + tile_size * tile_size;

  int tx = threadIdx.x, ty = threadIdx.y;

  int row = blockIdx.y * tile_size + ty;
  int col = blockIdx.x * tile_size + tx;

  float sum = 0;
  for (int tile = 0; tile < n / tile_size; ++tile) {
    shared_m1[ty * tile_size + tx] = m1[row * n + (tile * tile_size + tx)];
    shared_m2[ty * tile_size + tx] = m2[(tile * tile_size + ty) * n + col];
    __syncthreads();

    for (int k = 0; k < tile_size; ++k) {
      sum += shared_m1[ty * tile_size + k] * shared_m2[k * tile_size + tx];
    }
    __syncthreads();
  }

  if (row < n && col < n) {
    result[row * n + col] = sum;
  }
}

void measure_tiled(const vector<float> &m1, const vector<float> &m2, int size,
                   int runs, size_t tile_size) {
  // Transfer matrices to device
  float *d_m1, *d_m2, *d_result;
  checkCudaErrors(hipMalloc(&d_m1, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_m2, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_result, size * size * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_m1, m1.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));

  // Define grid and block dimensions
  dim3 blockDim(tile_size, tile_size);
  dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
               (size + blockDim.y - 1) / blockDim.y);
  size_t shared_mem_size = tile_size * tile_size * sizeof(float) * 2;

  if (size < 5000) {
    vector<float> result(size * size);

    // Run CUDA kernel
    tiled_matrix_multiply_kernel<<<gridDim, blockDim, shared_mem_size>>>(
        d_m1, d_m2, d_result, size, tile_size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    checkCudaErrors(hipMemcpy(result.data(), d_result,
                               size * size * sizeof(float),
                               hipMemcpyDeviceToHost));

    diff_with_cublas(m1, m2, size, result,
                     "tiled_cuda(tile_size=" + to_string(tile_size) + ")");
  }

  // Warmup
  for (int i = 0; i < 10; ++i) {
    tiled_matrix_multiply_kernel<<<gridDim, blockDim, shared_mem_size>>>(
        d_m1, d_m2, d_result, size, tile_size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }

  for (int i = 0; i < runs; ++i) {
    auto start_time = chrono::high_resolution_clock::now();
    tiled_matrix_multiply_kernel<<<gridDim, blockDim, shared_mem_size>>>(
        d_m1, d_m2, d_result, size, tile_size);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());
    auto end_time = chrono::high_resolution_clock::now();
    auto duration =
        chrono::duration_cast<chrono::microseconds>(end_time - start_time)
            .count();
    cout << size << "," << i + 1 << "," << duration
         << ",tiled_cuda(tile_size=" << tile_size << ")" << endl;
  }

  // Clean up
  checkCudaErrors(hipFree(d_m1));
  checkCudaErrors(hipFree(d_m2));
  checkCudaErrors(hipFree(d_result));
}

void measure_cublas(const vector<float> &m1, const vector<float> &m2, int size,
                    int runs) {
  hipblasHandle_t handle;
  checkCublasErrors(hipblasCreate(&handle));

  float *d_m1, *d_m2, *d_result;
  checkCudaErrors(hipMalloc(&d_m1, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_m2, size * size * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_result, size * size * sizeof(float)));
  checkCudaErrors(hipMemcpy(d_m1, m1.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2.data(), size * size * sizeof(float),
                             hipMemcpyHostToDevice));
  // Warmup
  for (int i = 0; i < 10; ++i) {
    cublas_matrix_multiply(handle, d_m1, d_m2, d_result, size);
  }

  for (int i = 0; i < runs; ++i) {
    auto start_time = chrono::high_resolution_clock::now();
    cublas_matrix_multiply(handle, d_m1, d_m2, d_result, size);
    auto end_time = chrono::high_resolution_clock::now();
    auto duration =
        chrono::duration_cast<chrono::microseconds>(end_time - start_time)
            .count();
    cout << size << "," << i + 1 << "," << duration << ",cublas" << endl;
  }

  hipblasDestroy(handle);
}

int main(int argc, char *argv[]) {
  if (argc != 3) {
    cerr << "Usage: " << argv[0] << " <size> <runs>\n";
    return EXIT_FAILURE;
  }

  try {
    int size = stoi(argv[1]);
    int runs = stoi(argv[2]);

    if (size <= 0) {
      throw invalid_argument("Size must be a positive integer.");
    }
    if (runs <= 0) {
      throw invalid_argument("Number of runs must be a positive integer.");
    }

    vector<float> m1 = create_random_matrix(size),
                  m2 = create_random_matrix(size);

    cout << "size,run,runtime_us,method" << endl;
    measure_naive(m1, m2, size, runs);
    measure_naive2(m1, m2, size, runs);
    measure_tiled(m1, m2, size, runs, 8);
    measure_tiled(m1, m2, size, runs, 16);
    measure_tiled(m1, m2, size, runs, 32);
    measure_cublas(m1, m2, size, runs);
  } catch (const exception &e) {
    cerr << "Error: " << e.what() << '\n';
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}