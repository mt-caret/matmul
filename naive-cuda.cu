
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <stdexcept>
#include <vector>

using namespace std;

vector<float> create_random_matrix(int n) {
  vector<float> matrix(n * n);
  for (int i = 0; i < n * n; ++i) {
    matrix[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
  }
  return matrix;
}

vector<float> naive_matrix_multiply(const vector<float> &m1,
                                    const vector<float> &m2, int n) {
  vector<float> result(n * n, 0);

  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      for (int k = 0; k < n; ++k) {
        result[i * n + j] += m1[i * n + k] * m2[k * n + j];
      }
    }
  }

  return result;
}

__global__ void naive_matrix_multiply_kernel(float *m1, float *m2,
                                             float *result, int n) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    float sum = 0;
    for (int k = 0; k < n; ++k) {
      sum += m1[row * n + k] * m2[k * n + col];
    }
    result[row * n + col] = sum;
  }
}

int main(int argc, char *argv[]) {
  if (argc != 3) {
    cerr << "Usage: " << argv[0] << " <size> <runs>\n";
    return EXIT_FAILURE;
  }

  try {
    int size = stoi(argv[1]);
    int runs = stoi(argv[2]);

    if (size <= 0) {
      throw invalid_argument("Size must be a positive integer.");
    }
    if (runs <= 0) {
      throw invalid_argument("Number of runs must be a positive integer.");
    }

    vector<float> m1 = create_random_matrix(size),
                  m2 = create_random_matrix(size);

    // Transfer matrices to device
    float *d_m1, *d_m2, *d_result;
    hipMalloc(&d_m1, size * size * sizeof(float));
    hipMalloc(&d_m2, size * size * sizeof(float));
    hipMalloc(&d_result, size * size * sizeof(float));
    hipMemcpy(d_m1, m1.data(), size * size * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2.data(), size * size * sizeof(float),
               hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(32, 32);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x,
                 (size + blockDim.y - 1) / blockDim.y);

    if (size < 1000) {
      // Check if lines up with CUDA implementation up to floating point error
      vector<float> naive_result = naive_matrix_multiply(m1, m2, size);
      vector<float> cuda_result(size * size);

      // Run CUDA kernel
      naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                          size);
      hipDeviceSynchronize();

      // Copy result back to host
      hipMemcpy(cuda_result.data(), d_result, size * size * sizeof(float),
                 hipMemcpyDeviceToHost);

      const float epsilon = 1e-3f;
      for (int i = 0; i < size * size; ++i) {
        if (std::abs(naive_result[i] - cuda_result[i]) > epsilon) {
          cerr << "Significant mismatch at index " << i
               << ": naive = " << naive_result[i]
               << ", cuda = " << cuda_result[i] << '\n';
          return EXIT_FAILURE;
        }
      }
    }

    // Warmup
    for (int i = 0; i < 10; ++i) {
      naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                          size);
      hipDeviceSynchronize();
    }

    cout << "size,run,runtime_us,method" << endl;
    for (int i = 0; i < runs; ++i) {
      auto start_time = chrono::high_resolution_clock::now();
      naive_matrix_multiply_kernel<<<gridDim, blockDim>>>(d_m1, d_m2, d_result,
                                                          size);
      hipDeviceSynchronize();
      auto end_time = chrono::high_resolution_clock::now();
      auto duration =
          chrono::duration_cast<chrono::microseconds>(end_time - start_time)
              .count();
      cout << size << "," << i + 1 << "," << duration << ",naive_cuda" << endl;
    }

    // Clean up
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_result);
  } catch (const exception &e) {
    cerr << "Error: " << e.what() << '\n';
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}